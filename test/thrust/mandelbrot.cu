#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/tabulate.h>
#include <thrust/execution_policy.h>
#include <thrust/copy.h>

#include <thrust/system/cuda/detail/synchronize.h>

#include <stdio.h>
#include <iostream>

#include "../../src/CycleTimer.h"

/*
 * Function taken from Assignment 1, part 3
 */
__host__ __device__
int mandel(float c_re, float c_im, int count) {
  float z_re = c_re, z_im = c_im;
  int i;
  for (i = 0; i < count; ++i) {

    if (z_re * z_re + z_im * z_im > 4.f)
      break;

    float new_re = z_re*z_re - z_im*z_im;
    float new_im = 2.f * z_re * z_im;
    z_re = c_re + new_re;
    z_im = c_im + new_im;
  }

  return i;
}

struct MandelIdxFunctor
{
  float x0;
  float y0;
  float dx;
  float dy;
  int   width;
  int   height;
  int   max_iters;

  MandelIdxFunctor(float _x0, float _y0, float _dx, float _dy,
      int _width, int _height, int _max_iters)
    : x0(_x0), y0(_y0), dx(_dx), dy(_dy),
      width(_width), height(_height), max_iters(_max_iters)
  {
    // empty
  }

  __host__ __device__
  int operator()(const int& i) const
  {
    float row = i / height;
    float col = i % width;

    float x = x0 + col * dx;
    float y = y0 + row * dy;

    return mandel(x, y, max_iters);
  }
};

void test_mandelbrot(int width=5000, int height=2000, int max_iters=250) {
  float x0 = -2;
  float x1 =  1;
  float y0 = -1;
  float y1 =  1;

  float dx = (x1 - x0) / width;
  float dy = (y1 - y0) / height;

  MandelIdxFunctor mandel_idx(x0, y0, dx, dy, width, height, max_iters);

  double start_time = CycleTimer::currentSeconds();
  thrust::device_vector<int> D(width * height);
  //thrust::host_vector<int>   H(width * height);

  thrust::tabulate(D.begin(), D.end(), mandel_idx);
  thrust::system::cuda::detail::synchronize();
  double end_time = CycleTimer::currentSeconds();

  printf("[mandelbrot parallel]:\t\t[%.3f] ms\n",
      (end_time - start_time) * 1000);

  //start_time = CycleTimer::currentSeconds();
  //thrust::tabulate(thrust::host, H.begin(), H.end(), mandel_idx);
  //end_time = CycleTimer::currentSeconds();

  //printf("[mandelbrot serial]:\t\t[%.3f] ms\n",
  //    (end_time - start_time) * 1000);

  // Ignore correctness checking because CUDA floating point is weird
  //for(int i = 0; i < width * height; i++) {
  //  //std::cout << i << ": " << D[i] << " " << H[i] << std::endl;
  //  if (D[i] != H[i]) {
  //    std::cout << "==> Not equal at " << i << ": " << D[i] << " " << H[i] << std::endl;
  //    return;
  //  }
  //}

}

int main(int argc, char *argv[]) {
  if (argc == 3) {
    int width     = atoi(argv[1]);
    int height    = atoi(argv[2]);

    test_mandelbrot(width, height);
  }
  else if (argc == 4) {
    int width     = atoi(argv[1]);
    int height    = atoi(argv[2]);
    int max_iters = atoi(argv[3]);

    test_mandelbrot(width, height, max_iters);
  }
  else if (argc != 1) {
    std::cout << "usage: " << argv[0]
      << " [width height [max_iters]]" << std::endl;

    return 1;
  }
  else {
    test_mandelbrot();
  }
}
