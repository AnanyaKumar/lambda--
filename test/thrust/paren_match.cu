#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include <thrust/tabulate.h>

#include <thrust/system/cuda/detail/synchronize.h>

#include <limits>
#include <stdio.h>
#include <iostream>

#include "../../src/CycleTimer.h"

struct Generator1 {
  __host__ __device__
  int operator()(const int& i) const {
    return i % 2 == 0 ? 1 : -1;
  }
};
struct Generator2 {
  const int n;
  Generator2(int length) : n(length) {}

  __host__ __device__
  int operator()(const int& i) const {
    return i < n / 2 ? 1 : -1;
  }
};
struct Generator3 {
  __host__ __device__
  int operator()(const int& i) const {
    return i % 2 == 0 ? -1 : 1;
  }
};
struct Generator4 {
  int n;
  Generator4(int length) : n(length) {}

  __host__ __device__
  int operator()(const int& i) const {
    return i < n / 2 ? -1 : 1;
  }
};

bool paren_match(thrust::device_vector<int> &D) {
  double start_time = CycleTimer::currentSeconds();
  thrust::exclusive_scan(D.begin(), D.end(), D.begin());

  int int_max = std::numeric_limits<int>::max();
  int min = thrust::reduce(D.begin(), D.end(), int_max, thrust::minimum<int>());

  bool result = D[D.size() - 1] == 0 && min >= 0;
  double end_time = CycleTimer::currentSeconds();

  printf("[paren_match parallel]:\t\t[%.3f] ms\n",
      (end_time - start_time) * 1000);

  return result;
}

void test_paren_match(int length=10000) {
  thrust::device_vector<int> D1(length);
  thrust::device_vector<int> D2(length);
  thrust::device_vector<int> D3(length);
  thrust::device_vector<int> D4(length);

  thrust::tabulate(D1.begin(), D1.end(), Generator1());
  thrust::tabulate(D2.begin(), D2.end(), Generator2(length));
  thrust::tabulate(D3.begin(), D3.end(), Generator3());
  thrust::tabulate(D4.begin(), D4.end(), Generator4(length));

  paren_match(D1);
  thrust::system::cuda::detail::synchronize();

  paren_match(D2);
  thrust::system::cuda::detail::synchronize();

  paren_match(D3);
  thrust::system::cuda::detail::synchronize();

  paren_match(D4);
  thrust::system::cuda::detail::synchronize();

}

int main(int argc, char *argv[]) {
  if (argc == 2) {
    int length = atoi(argv[1]);

    test_paren_match(length);
  }
  else if (argc != 1) {
    std::cout << "usage: " << argv[0] << " [length]" << std::endl;

    return 1;
  }
  else {
    test_paren_match();
  }
}
